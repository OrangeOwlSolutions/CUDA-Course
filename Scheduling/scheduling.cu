#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

/*******************************/
/* STREAMING MULTIPROCESSOR ID */
/*******************************/
static __device__ __inline__ uint32_t __mysmid() {
	uint32_t smid;
	asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
	return smid;
}

/***********/
/* WARP ID */
/***********/
static __device__ __inline__ uint32_t __mywarpid() {
	uint32_t warpid;
	asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
	return warpid;
}

/****************/
/* WARP LANE ID */
/****************/
static __device__ __inline__ uint32_t __mylaneid() {
	uint32_t laneid;
	asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));
	return laneid;
}

/*******************/
/* KERNEL FUNCTION */
/*******************/
__global__ void mykernel() {

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	
	printf("Thread ID = %d;\t SM ID = %d;\t Warp ID = %d;\t Warp lane = %d\n", idx, __mysmid(), __mywarpid(), __mylaneid());
}

/********/
/* MAIN */
/********/
int main() {

	mykernel << <2, 64 >> >();
	hipDeviceSynchronize();

	return 0;

}
